#include "hip/hip_runtime.h"
#include "cudaDoseCalc.h"
#include "cudaInit.h"
#include "kernel.h"
#include "brain_defs.h"
#include "configure.h"
#include "binary_io.h"
#include "geometry.h"
#include "debugLog.h"

#include <iostream>
#include <string>
#include <iomanip>
#include "hip/hip_runtime.h"
#include "boost/filesystem.hpp"

namespace fs = boost::filesystem;

int old::radconvolveTexture (
    MONO_KERNELS        *mono,
    CONSTANTS           *constants,
    std::vector<BEAM>&  beams,
    int                 nrays,
    RES_LOG& result)
{
    float3 rev_voxelsize = {constants->rev_latspacing, 
        constants->rev_longspacing, constants->rev_latspacing};

    // Set up arrays for dynamic GPU resource allocation
    // depending on dimensions of BEV data per convolution ray
    dim3 tileBlock(TILE_DIM_X, TILE_DIM_Y, 1);
    std::vector<dim3> rayGrid(nrays);
    std::vector<dim3> conBlock(nrays);
    std::vector<dim3> conGrid(nrays);
    std::vector<uint> memsize(nrays);

    // Calculate cuda execution block/grid sizes
    uint dataSize = constants->nvoxels();
    uint calcDataSize = constants->bbox_nvoxels();

    uint3 max_actual_rev_size = {0};
    for (int dc=0; dc<beams.size(); dc++)
    {
        BEAM& this_beam = beams[dc];
        std::cout << "Starting: | beam#" << dc << std::endl;

        // 1. initialize fluence map. Here, we enable all beamlets
        float* d_fluence_map;
        int fluenceSize = this_beam.fmap_size.x * this_beam.fmap_size.y;
        std::vector<float> h_fluence_map(fluenceSize, 1.);
        checkCudaErrors(hipMalloc((void**)(&d_fluence_map), fluenceSize * sizeof(float)));
        checkCudaErrors(hipMemcpy(d_fluence_map, h_fluence_map.data(), 
            fluenceSize*sizeof(float), hipMemcpyHostToDevice));

        PILLAR_GRID hPG{};
        hPG.numBeamlets = this_beam.fmap_size.x * this_beam.fmap_size.y;
        hPG.beamletIdx = std::vector<int>(hPG.numBeamlets, 0);
        hPG.pillarStartCoords = std::vector<float3>(hPG.numBeamlets);
        hPG.beamletAngles = std::vector<float2>(hPG.numBeamlets);
        hPG.beamletIsocenters = std::vector<float3>(hPG.numBeamlets);

        for (int i=0; i<hPG.numBeamlets; i++)
            hPG.beamletIdx[i] = i;
        
        // 2. get the central axis limit (anchors) of beamlets
        std::vector<float3> beamlet_start(hPG.numBeamlets);
        std::vector<float3> beamlet_end(hPG.numBeamlets);
        std::vector<float> beamlet_length(hPG.numBeamlets);
        float max_beamlet_length = 1.0f;
        hPG.max_beamlet_size = float2{0.f, 0.f};
        for (int i=0; i<hPG.numBeamlets; i++)
        {
            calcBeamletAnchors(
                beamlet_start[i], beamlet_end[i], hPG.beamletAngles[i], hPG.beamletIsocenters[i],
                this_beam.source, this_beam.isocenter,
                static_cast<uint>(hPG.beamletIdx[i]),
                this_beam.beamlet_size, this_beam.fmap_size,
                constants->voxel,
                constants->start,
                constants->calc_bbox_start,
                constants->calc_bbox_size,
                this_beam.azimuth, this_beam.zenith, this_beam.coll
            );
            beamlet_length[i] = length(beamlet_end[i] - beamlet_start[i]);
            max_beamlet_length = std::max(max_beamlet_length, beamlet_length[i]);
            float2 beamlet_diverge_size = this_beam.beamlet_size * 
                length(beamlet_end[i]-this_beam.source) /
                length(hPG.beamletIsocenters[i] - this_beam.source);
            hPG.max_beamlet_size.x = max(hPG.max_beamlet_size.x, beamlet_diverge_size.x);
            hPG.max_beamlet_size.y = max(hPG.max_beamlet_size.y, beamlet_diverge_size.y);
        }

        // compute pillar size and ensure that it is an integer multiple of rev_voxelsize
        float psize_long = max_beamlet_length + 2. * constants->kernel_extent + hPG.wallThickness*rev_voxelsize.y;
        float2 psize_trans = hPG.max_beamlet_size + 2. * constants->kernel_extent + hPG.wallThickness*rev_voxelsize.x;
        float3 expand = rev_voxelsize - make_float3(
            fmodf(psize_trans.x, rev_voxelsize.x),
            fmodf(psize_long,    rev_voxelsize.y),
            fmodf(psize_trans.y, rev_voxelsize.z));
        hPG.pillarSize = make_float3(psize_trans.x, psize_long, psize_trans.y) + 
            expand + 2.*float(hPG.pillarBuffer)*rev_voxelsize;
        hPG.pillarDims = make_int3(hPG.pillarSize / rev_voxelsize);

        // compute pillar limits to use in geometric transformations
        for (int i=0; i<hPG.numBeamlets; i++)
        {
            float3 g_offset = make_float3(
                -0.5f * (hPG.pillarSize.x + hPG.wallThickness*rev_voxelsize.x),
                -0.5f * (hPG.pillarSize.y - beamlet_length[i] + hPG.wallThickness*rev_voxelsize.y),
                -0.5f * (hPG.pillarSize.z + hPG.wallThickness*rev_voxelsize.z) );
            // TODO fix rotation pivot to src here (may be unnecessary)
            g_offset = inverseRotateBeamAtOriginRHS(
                    inverseRotateBeamletAtOriginRHS(g_offset, hPG.beamletAngles[i].x, hPG.beamletAngles[i].y),
                        this_beam.azimuth, this_beam.zenith, this_beam.coll);
            hPG.pillarStartCoords[i] = beamlet_start[i] + g_offset;
        }

        // 3. Build the packed BEV allocation map
        hPG.numPillars.x = static_cast<int>(std::ceil(std::sqrt(float(hPG.numBeamlets))));
        hPG.numPillars.y = static_cast<int>(std::ceil(float(hPG.numBeamlets) / hPG.numPillars.x));

        // add another wall at the end of the array in each lateral dimension
        hPG.gridDims = make_int3(
            hPG.numPillars.x * hPG.pillarDims.x + hPG.wallThickness,
            hPG.pillarDims.y + hPG.wallThickness,
            hPG.numPillars.y * hPG.pillarDims.z + hPG.wallThickness
        );
        hPG.gridSize = make_float3(hPG.gridDims) * rev_voxelsize;

        int*    dpg_beamletIdx;
        float3* dpg_pillarStartCoords;
        float2* dpg_beamletAngles;
        float3* dpg_beamletIsocenters;
        checkCudaErrors(hipMalloc((void**)(&dpg_beamletIdx), hPG.numBeamlets*sizeof(int)));
        checkCudaErrors(hipMalloc((void**)(&dpg_pillarStartCoords), hPG.numBeamlets*sizeof(float3)));
        checkCudaErrors(hipMalloc((void**)(&dpg_beamletAngles), hPG.numBeamlets*sizeof(float2)));
        checkCudaErrors(hipMalloc((void**)(&dpg_beamletIsocenters), hPG.numBeamlets*sizeof(float3)));
        checkCudaErrors(hipMalloc((void**)(&(device_data.dose)), hPG.pillar_grid_nvoxels()*sizeof(float)));

        // Move pillar_grid data to device
        checkCudaErrors(hipMemcpy(dpg_beamletIdx, hPG.beamletIdx.data(), hPG.numBeamlets*sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dpg_pillarStartCoords, hPG.pillarStartCoords.data(), hPG.numBeamlets*sizeof(float3), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dpg_beamletAngles, hPG.beamletAngles.data(), hPG.numBeamlets*sizeof(float2), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dpg_beamletIsocenters, hPG.beamletIsocenters.data(), hPG.numBeamlets*sizeof(float3), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemset(device_data.dose, 0., hPG.pillar_grid_nvoxels()*sizeof(float)));

        dim3 packedGrid = dim3(
            static_cast<uint>(std::ceil(static_cast<float>(hPG.gridDims.y) / tileBlock.x)),
            static_cast<uint>(std::ceil(static_cast<float>(hPG.gridDims.z) / tileBlock.y)),
            static_cast<uint>(std::ceil(static_cast<float>(hPG.gridDims.x) / tileBlock.z))
        );

        dim3 rayBlock(1, 32, 3);
        std::vector<REV_DATA> rev(nrays);

        // compute
        radconvolvePrep(constants, &hPG,
            nrays, rev, this_beam, rayGrid, rayBlock,
            conGrid, conBlock, memsize, max_actual_rev_size);

        radconvolveCompute(
            mono, constants, nrays, this_beam,
            hPG, dpg_beamletIdx, dpg_beamletAngles,
            dpg_pillarStartCoords, dpg_beamletIsocenters,
            rayGrid, rayBlock,
            rev, d_fluence_map,
            conGrid, conBlock,
            memsize, packedGrid, tileBlock,
            dc, result[dc]
        );

        // clean up
        checkCudaErrors(hipFree(device_data.dose));
        checkCudaErrors(hipFree(dpg_beamletIsocenters));
        checkCudaErrors(hipFree(dpg_beamletAngles));
        checkCudaErrors(hipFree(dpg_pillarStartCoords));
        checkCudaErrors(hipFree(dpg_beamletIdx));
        checkCudaErrors(hipFree(d_fluence_map));
    }
    return 0;
}

int old::radconvolvePrep(CONSTANTS* constants, PILLAR_GRID* hPG, 
    int nrays, std::vector<REV_DATA>& rev, BEAM& this_beam, std::vector<dim3>& rayGrid, dim3& rayBlock,
    std::vector<dim3>& conGrid, std::vector<dim3>& conBlock, std::vector<uint>& memsize,
    uint3& max_actual_rev_size)
{
    for (int rr=0; rr<nrays; rr++)
    {
        float theta = constants->get_theta_from_index(rr);
        float phi = constants->get_phi_from_index(rr);

        findREV(&rev[rr],
            constants,
            &this_beam,
            make_float3(0.f),
            hPG->gridSize,
            theta,
            phi);
        
        // PAY ATTENTION: rev[rr].size in the order of YZX
        // rayGrid[rr] in the order of XYZ
        rayGrid[rr] = make_uint3(rev[rr].size.z,
            static_cast<uint>(std::ceil(static_cast<float>(rev[rr].size.x) / rayBlock.y)),
            static_cast<uint>(std::ceil(static_cast<float>(rev[rr].size.y) / rayBlock.z)));
        
        // rev[rr].size in the order of YZX
        if (rev[rr].size.x > 1024)
            throw std::runtime_error("size of rev (x-dim) exceeds maximum #threads allowed. Contact dev to resolve.");
        
        // buffer beyond used rev block, where convolve kernel 
        // will also write zero before proceeding
        // conBlock in the order of YZX
        int overwrite = 2;
        conBlock[rr].x = rev[rr].size.x + overwrite;
        conGrid[rr].y  = rev[rr].size.y + overwrite;
        conGrid[rr].z  = rev[rr].size.z + overwrite;
        conBlock[rr].y = conBlock[rr].z = conGrid[rr].x = 1;
        memsize[rr] = 2 * (conBlock[rr].x - overwrite) * sizeof(float);

        max_actual_rev_size.x = std::max(max_actual_rev_size.x, rev[rr].size.x);
        max_actual_rev_size.y = std::max(max_actual_rev_size.y, rev[rr].size.y);
        max_actual_rev_size.z = std::max(max_actual_rev_size.z ,rev[rr].size.z);
    }
    return 0;
}

int old::radconvolveCompute(
    MONO_KERNELS *mono, CONSTANTS* constants, int nrays, BEAM& this_beam,
    PILLAR_GRID& hPG, int* dpg_beamletIdx, float2* dpg_beamletAngles,
    float3* dpg_pillarStartCoords, float3* dpg_beamletIsocenters,
    const std::vector<dim3>& rayGrid, const dim3& rayBlock,
    const std::vector<REV_DATA>& rev, float* d_fluence_map,
    const std::vector<dim3>& conGrid, const std::vector<dim3>& conBlock, 
    const std::vector<uint>& memsize, const dim3& packedGrid, const dim3& tileBlock,
    int dc, BEAM_LOG& beamResult)
{
    float3 rev_voxelsize = {constants->rev_latspacing, constants->rev_longspacing, constants->rev_latspacing};
    fs::path debugDir(Paths::Instance()->debug_dir());
    for (int ray_idx=0; ray_idx<nrays; ray_idx++)
    {
        // get kernel rotation angles once instead of for each kernel thread launch
        int kern_wt_idx = constants->get_kernel_theta_index(ray_idx);
        float kern_theta = constants->get_theta_from_index(ray_idx);
        float kern_phi = constants->get_phi_from_index(ray_idx);

        // calculate rev Terma and sample density
        // rayGrid[rr] in the order of XYZ
        int raySharedMem = rayBlock.x * rayBlock.y * rayBlock.z * sizeof(float);

        float3* g_coords_log = nullptr;
        if (constants->debugREVTerma && dc==0 && ray_idx==0)
        {
            #include "debug_raytrace.cpp.in"
            int volume = constants->max_rev_size.x * 
                constants->max_rev_size.y * constants->max_rev_size.z;
            checkCudaErrors(hipMalloc((void**)(&g_coords_log), volume*sizeof(float3)));
        }

        cudaBeamletRaytrace<<<rayGrid[ray_idx], rayBlock, raySharedMem>>>(
            device_data.revDens,
            device_data.revTerma,
            this_beam.source,
            this_beam.beamlet_size,
            this_beam.azimuth,
            this_beam.zenith,
            this_beam.coll,
            make_float3(hPG.gridDims),
            make_float3(hPG.pillarDims),
            hPG.numBeamlets,
            hPG.wallThickness,
            hPG.numPillars,
            dpg_beamletIdx,
            dpg_beamletAngles,
            dpg_pillarStartCoords,
            dpg_beamletIsocenters,
            kern_theta, kern_phi,
            rev[ray_idx].min_coords,
            rev[ray_idx].size,
            constants->max_rev_size,
            constants->start,
            constants->voxel,
            rev_voxelsize,
            make_float3(constants->calc_bbox_start),
            texDens,
            g_coords_log,

            // raytrcing/terma args
            d_fluence_map,
            make_float3(constants->size),
            constants->calc_bbox_size,
            constants->beamhard_correct,
            this_beam.direction,
            this_beam.isocenter,
            this_beam.sad,
            this_beam.fmap_size,
            mono->nkernels,
            texSpectrum
        );

        float* d_debugProbe=nullptr;
        if (constants->debugREVTerma && dc==0 && ray_idx==0)
        {
            if (write_g_coords_log(g_coords_log, constants, debugDir))
                return 1;
            checkCudaErrors(hipFree(g_coords_log));
            if (writeREVTerma(device_data.revTerma, 
                device_data.revDens, constants, debugDir))
                return 1;
            #include "debug_convolve.cpp.in"
        }
        if (constants->debugREVDose && dc==0 && ray_idx==0)
        {
            int volume = constants->max_rev_size.x * 
                constants->max_rev_size.y * constants->max_rev_size.z;
            checkCudaErrors(hipMalloc((void**)(&d_debugProbe), volume*sizeof(float)));
            checkCudaErrors(hipMemset(d_debugProbe, 0, volume*sizeof(float)));
        }

        // perform dose calculation (CCCS) w/ heterogeneity correction in REV volume
        PackRowConvolve<<<conGrid[ray_idx], conBlock[ray_idx], memsize[ray_idx]>>> (
            device_data.revDens,
            device_data.revTerma,
            device_data.surfDose,
            (float)kern_wt_idx,
            rev[ray_idx].size,
            constants->max_rev_size,
            constants->rev_longspacing,
            constants->nradii,
            constants->ntheta,
            constants->nphi,
            texKern,
            d_debugProbe
        );

        if (constants->debugREVDose && dc==0 && ray_idx==0)
        {
            if (writeREVDose(device_data.texDose, constants, debugDir))
                return 1;
            if (writeREVDebug(d_debugProbe, constants, debugDir))
                return 1;
            if (writeREVSurf(device_data.surfDose, constants, debugDir))
                return 1;
            checkCudaErrors(hipFree(d_debugProbe));
        }

        // transform packed REV dose coefficients from the previous convolution back to the BEV system then
        // perform element-by-element sum, accumulating over all convolution directions
        PackedREVtoBEVdose <<<packedGrid, tileBlock, 0>>> (
            device_data.dose,             // beamlet-packed dose array in BEV orientation
            device_data.texDose,          // packed dose array embedded in REV bounding box
            kern_theta, kern_phi,         // convolution direction
            rev[ray_idx].min_coords,      // REV volume limit coords in XYZ coord system
            rev_voxelsize,
            hPG.gridDims
        );

        // for debug purposes
        if (constants->debugREVDose)
            std::cout << "ray index: " << ray_idx << std::endl;
    }

    if (constants->debugBEVDose && dc==0)
        if (writeBEVDose(device_data.dose, hPG, debugDir))
            return 1;
    
    // if (! constants->logPatientDose)
    //     return 0;

    // unpack pillars from BEV storage
    // copy output of PackedREVtoBEVdose to hipArray and attach texture object
    hipChannelFormatDesc floatChannelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipExtent packedArrayExtent = make_hipExtent(hPG.gridDims.y, hPG.gridDims.z, hPG.gridDims.x);
    hipArray_t PackedBEVdose_Array;
    checkCudaErrors(hipMalloc3DArray(&PackedBEVdose_Array, &floatChannelDesc, packedArrayExtent));
    // copy to hipArray
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)device_data.dose, 
        packedArrayExtent.width*sizeof(float), packedArrayExtent.width, 
        packedArrayExtent.height);
    copyParams.dstArray = PackedBEVdose_Array;
    copyParams.extent = packedArrayExtent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3DAsync(&copyParams));
    // attach to texture object
    hipTextureObject_t texPackedBEVDose;
    makeTexObject(&texPackedBEVDose, PackedBEVdose_Array, 3, hipAddressModeBorder, hipFilterModeLinear);

    // determine kernel launch params
    dim3 unpackBlock = tileBlock;
    dim3 unpackGrid = dim3(
        static_cast<uint>(std::ceil(static_cast<float>(constants->calc_bbox_size.x) / unpackBlock.x)),
        static_cast<uint>(std::ceil(static_cast<float>(constants->calc_bbox_size.y) / unpackBlock.y)),
        static_cast<uint>(std::ceil(static_cast<float>(constants->calc_bbox_size.z) / unpackBlock.z))
    );

    int calcDataSize = constants->bbox_nvoxels();
    std::get<1>(beamResult).resize(hPG.numBeamlets);
    float* d_unpacked_dose;
    checkCudaErrors(hipMalloc((void**)(&d_unpacked_dose), calcDataSize*sizeof(float)));
    for (int i=0; i<hPG.numBeamlets; i++)
    {
        std::get<1>(beamResult)[i].resize(calcDataSize);
        checkCudaErrors(hipMemset(d_unpacked_dose, 0., calcDataSize*sizeof(float)));
        UnpackBEVDosePillar <<<unpackGrid, unpackBlock>>> (
            d_unpacked_dose,
            texPackedBEVDose,
            this_beam.sad,
            this_beam.source,
            this_beam.azimuth,
            this_beam.zenith,
            this_beam.coll,
            constants->start,
            constants->voxel,
            make_float3(constants->calc_bbox_start),
            constants->calc_bbox_size,
            rev_voxelsize,
            i,
            i % hPG.numPillars.x,
            i / hPG.numPillars.x,
            hPG.pillarDims,
            hPG.wallThickness,
            hPG.pillarBuffer,
            hPG.pillarStartCoords[i],
            hPG.beamletAngles[i]
        );
        checkCudaErrors(hipMemcpy(std::get<1>(beamResult)[i].data(), d_unpacked_dose, 
            calcDataSize*sizeof(float), hipMemcpyDeviceToHost));
    }
    std::get<0>(beamResult) = hPG;
    checkCudaErrors(hipFree(d_unpacked_dose));
    checkCudaErrors(hipDestroyTextureObject(texPackedBEVDose));
    checkCudaErrors(hipFreeArray(PackedBEVdose_Array));

    return 0;
}